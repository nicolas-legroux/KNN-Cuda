#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#include <helper_functions.h>
#include <stdlib.h>
#include <stdio.h>
#include "../configuration.h"
#include "compute_distances.h"
#include "../utilities.h"

#define MAX_THREADS_PER_BLOCK 1024
#define SIMPLE_BLOCK_SIZE 1024
#define BLOCK_DIM 32

__global__ void gpu_distance_withreduction(double* data, double* distance,
		double* point, int n, int dim) {

	extern __shared__ double distComponents[];

	int shift_dim = threadIdx.x;
	int shift_point = blockIdx.x * blockDim.y + threadIdx.y;
	int shift_point_in_block = threadIdx.y;

	if (shift_dim < DIM && shift_point < n) {
		double d = 0;
		d = abs(data[shift_point * dim + shift_dim] - point[shift_dim]);
		distComponents[shift_point_in_block * blockDim.y + shift_dim] = d * d;
	}

	__syncthreads();

	for (int offset = blockDim.x / 2; offset > 0; offset >>= 1) {

		if (shift_dim < offset) {
			distComponents[shift_point_in_block * blockDim.y + shift_dim] +=
					distComponents[shift_point_in_block * blockDim.y + shift_dim
							+ offset];
		}

		__syncthreads();
	}

	if (shift_dim == 0 && shift_point < n) {
		distance[shift_point] =
				distComponents[shift_point_in_block * blockDim.y];
	}
}

__global__ void gpu_distance(double* data, double* distance, double* point,
		int n, int dim) {
	int i = blockIdx.x * blockDim.x + threadIdx.x;

	if (i >= n)
		return;

	double d = 0;

	for (int j = 0; j < dim; j++) {
		double temp = abs(data[i * dim + j] - point[j]);
		d += temp * temp;
	}

	distance[i] = d;
}

//blockDim.x must be larger than Dim
//BlockDim.y should be equal to one
//GridDim.y should be equal to the number of training points
//test_data should be stored in row-major order
//train_data should be stored in column-major order
__global__ void gpu_distances(double* train_data, double * test_data, int dim, int n_train, int n_test, double * distances){

	int t_idx_in_block = threadIdx.x;
	int t_idx_global = blockDim.x * blockIdx.x + threadIdx.x;
	int t_idy_global = blockIdx.y;

	extern __shared__ double test_data_point[];

	//Load one data point into shared memory
	if(t_idx_in_block < dim){
		test_data_point[t_idx_in_block] = test_data[t_idy_global*dim + t_idx_in_block];
	}

	__syncthreads();

	//Now compute distance
	double dist = 0;
	if(t_idx_global < n_train){
		for(int i=0; i<dim; i++){
			double temp = train_data[i * n_train + t_idx_global] - test_data_point[i];
			dist += temp*temp;
		}

		distances[t_idy_global*n_train + t_idx_global] = dist;
	}
}

//All The data is assumed to be in row major order
/* At the end the distance matrix is as follows :
 *  [ distance(test1, train1)    distance(test2, train1)   distances(test3, train1) ...
 *   distance(test1, train2     ...
 *  ...																					]
 */
void gpu_compute_distances(double *train_data, double *test_data, int n_train, int n_test, int dim, double* distances){
	double *train_data_copy = new double[n_train*dim];
	array_copy(train_data, train_data_copy, n_train*dim);
	convert_row_major_to_column_major(train_data_copy, n_train, dim);

	double* d_train_data;
	double* d_test_data;
	double* d_distances;

	checkCudaErrors(hipMalloc((void**)&d_train_data, n_train*dim*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d_test_data, n_test*dim*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d_distances, n_train*n_test*sizeof(double)));

	checkCudaErrors(hipMemcpy(d_train_data, train_data_copy, n_train*dim*sizeof(double), hipMemcpyHostToDevice));
	checkCudaErrors(hipMemcpy(d_test_data, test_data, n_test*dim*sizeof(double), hipMemcpyHostToDevice));

	int dim_grid_y = n_test;
	int dim_block_x = multiple_of_32(dim);
	int dim_block_y = 1;
	int dim_grid_x = n_train/dim_block_x;

	if(n_train%dim_block_x != 0)
		dim_grid_x++;

	dim3 dim_grid(dim_grid_x, dim_grid_y);
	dim3 dim_block(dim_block_x, dim_block_y);

	gpu_distances<<<dim_grid, dim_block, dim*sizeof(double)>>>(d_train_data, d_test_data, dim, n_train, n_test, d_distances);

	checkCudaErrors(
			hipMemcpy(distances, d_distances, n_train*n_test*sizeof(double), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_train_data));
	checkCudaErrors(hipFree(d_test_data));
	checkCudaErrors(hipFree(d_distances));

	delete[] train_data_copy;
}

void gpu_compute_distance(double* data, double* point, double* distance) {

	int datasize = N * DIM * sizeof(double);

	int nblock = N / SIMPLE_BLOCK_SIZE;
	if (N % SIMPLE_BLOCK_SIZE != 0)
		nblock += 1;
	int nthread = SIMPLE_BLOCK_SIZE;

	double *d_data;
	double *d_point;
	double *d_distance;

	printf("\nGrid dimension : %d\n", nblock);
	printf("Block dimension : %d\n", nthread);

	checkCudaErrors(hipMalloc((void**)&d_data, datasize));
	checkCudaErrors(hipMalloc((void**)&d_distance, N*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d_point, DIM*sizeof(double)));

	checkCudaErrors(
			hipMemcpy(d_data, data, datasize, hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(d_point, point, DIM*sizeof(double), hipMemcpyHostToDevice));

	gpu_distance<<<nblock, nthread>>>(d_data, d_distance, d_point, N, DIM);

	checkCudaErrors(
			hipMemcpy(distance, d_distance, N*sizeof(double), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_distance));
	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(d_point));
}



void gpu_compute_distance_withreduction(double* data, double* point,
		double* distance) {
	int datasize = N * DIM * sizeof(double);

	int block_dim_x = multiple_of_32(DIM);
	int block_dim_y = MAX_THREADS_PER_BLOCK / block_dim_x;

	int nblock = N / block_dim_y;
	if (N % block_dim_y != 0)
		nblock++;

	dim3 dim_block(block_dim_x, block_dim_y, 1);

	printf("\nGrid dimension : %d\n", nblock);
	printf("Block dimension : %d * %d\n", dim_block.x, dim_block.y);

	double *d_data;
	double *d_point;
	double *d_distance;

	checkCudaErrors(hipMalloc((void**)&d_data, datasize));
	checkCudaErrors(hipMalloc((void**)&d_distance, N*sizeof(double)));
	checkCudaErrors(hipMalloc((void**)&d_point, DIM*sizeof(double)));

	checkCudaErrors(
			hipMemcpy(d_data, data, datasize, hipMemcpyHostToDevice));
	checkCudaErrors(
			hipMemcpy(d_point, point, DIM*sizeof(double), hipMemcpyHostToDevice));

	gpu_distance_withreduction<<<nblock, dim_block,
			block_dim_y * block_dim_x * sizeof(double)>>>(d_data, d_distance,
			d_point, N, DIM);

	checkCudaErrors(
			hipMemcpy(distance, d_distance, N*sizeof(double), hipMemcpyDeviceToHost));

	checkCudaErrors(hipFree(d_distance));
	checkCudaErrors(hipFree(d_data));
	checkCudaErrors(hipFree(d_point));
}

/*

 int gpu_knn(int * cdata_c, int * data_c, int * point_c, int nclass) {

 int datasize = N * DIM * sizeof(int);

 int nblock = N / blocksize, nthread = blocksize;

 double *distance = new double[N];

 int *d_data;
 int *d_point;
 double *d_distance;

 checkCudaErrors(hipMalloc((void**)&d_data, datasize));
 checkCudaErrors(hipMalloc((void**)&d_distance, N*sizeof(double)));
 checkCudaErrors(hipMalloc((void**)&d_point, DIM*sizeof(int)));

 checkCudaErrors(
 hipMemcpy(d_data, data_c, datasize, hipMemcpyHostToDevice));
 checkCudaErrors(
 hipMemcpy(d_point, point_c, DIM*sizeof(int), hipMemcpyHostToDevice));

 gpu_distance<<<nblock, nthread>>>(d_data, d_distance, d_point, N, DIM);

 checkCudaErrors(
 hipMemcpy(distance, d_distance, N*sizeof(double), hipMemcpyDeviceToHost));

 checkCudaErrors(hipFree(d_distance));
 checkCudaErrors(hipFree(d_data));
 checkCudaErrors(hipFree(d_point));

 return -1;
 }

 */

